#include "hip/hip_runtime.h"
#include "nearest_neighbour.cuh"

#include <cmath>

#define NREPS 10 // number of repetations for time calculations

#define THREADS_PER_BLOCK 1024

__global__ void NearestNeighbourKernel(Point *train, Point *test, int *result, int trainSize, int testSize)
{
	//int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	//unsigned int i = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int blockId = blockIdx.y * gridDim.x + blockIdx.x;
	int i = blockId * blockDim.x + threadIdx.x;
	if (i < testSize)
	{
		__shared__ int minDist;
		__shared__ int minID;
		__shared__ int dist;
		__shared__ uint4s minMax;

		minDist = INT32_MAX;
		minID = -1;

		for (int j = 0; j < trainSize; j++)
		{
			dist = 0;
			// Calculate distance between points
			for (int k = 0; k < 8; k++)
			{
				// calculate max-min of 2 numbers without branching (hack)
				minMax.first = train[j][k].first ^ ((test[i][k].first ^ train[j][k].first) & -(test[i][k].first < train[j][k].first)); // min(x, y)
				minMax.second = test[i][k].first ^ ((test[i][k].first ^ train[j][k].first) & -(test[i][k].first < train[j][k].first)); // max(x, y)
				dist += (minMax.second - minMax.first) * (minMax.second - minMax.first); // (max(x,y)-min(x,y))^2

				minMax.first = train[j][k].second ^ ((test[i][k].second ^ train[j][k].second) & -(test[i][k].second < train[j][k].second)); // min(x, y)
				minMax.second = test[i][k].second ^ ((test[i][k].second ^ train[j][k].second) & -(test[i][k].second < train[j][k].second)); // max(x, y)
				dist += (minMax.second - minMax.first) * (minMax.second - minMax.first); // (max(x,y)-min(x,y))^2
			}
			if (dist < minDist)
			{
				minDist = dist;
				minID = j;
			}
		}
		result[i] = minID;
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t CudaNearestNeighbour(Point *train, Point *test, int *result, int trainSize, int testSize)
{
	Point *dev_train, *dev_test;
	int *dev_result;
	int *dev_trainSize, *dev_testSize;
	hipError_t cudaStatus;

	int numThreads = (int)sqrt(THREADS_PER_BLOCK);
	dim3 dimBlock(numThreads, numThreads, 1);

	//===========================================================================================================================
	// Allocate GPU buffers for three vectors (two input, one output)
	//
	cudaStatus = hipMalloc((void**)&dev_result, testSize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_train, trainSize * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_test, testSize * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	cudaStatus = hipMalloc((void**)&dev_testSize, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_trainSize, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//===========================================================================================================================
	// Copy input vectors from host memory to GPU buffers.
	//
	cudaStatus = hipMemcpy(dev_train, train, trainSize * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_test, test, testSize * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	
	cudaStatus = hipMemcpy(dev_testSize, &testSize, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_trainSize, &trainSize, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//===========================================================================================================================
	// Launch a kernel on the GPU with one thread for each element, and check for errors.
	//
	NearestNeighbourKernel<<<(testSize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, dimBlock>>>(dev_train, dev_test, dev_result, *dev_trainSize, *dev_testSize);

	//===========================================================================================================================
	// Check for any errors launching the kernel
	//
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, dev_result, testSize * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_result);
	hipFree(dev_train);
	hipFree(dev_test);
	hipFree(dev_testSize);
	hipFree(dev_trainSize);

	return cudaStatus;
}
